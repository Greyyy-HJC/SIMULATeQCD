#include "hip/hip_runtime.h"
#include "integrator.h"


// inline void WaitEnter() { rootLogger.warn("Press Enter to continue..."); while (std::cin.get()!='\n'); }

template<class floatT, size_t HaloDepth, CompressionType comp=R18>
struct do_evolve_Q
{
    do_evolve_Q(gaugeAccessor<floatT, comp> gAcc,gaugeAccessor<floatT> pAccessor,floatT stepsize) : _stepsize(stepsize),
    _pAccessor(pAccessor), _gAcc(gAcc){}

    
    double _stepsize;
    gaugeAccessor<floatT> _pAccessor;
    gaugeAccessor<floatT, comp> _gAcc;

    __device__ __host__ GSU3<floatT> operator()(gSiteMu site){
        typedef GIndexer<All,HaloDepth> GInd;

        GSU3<double> temp;

        temp= su3_exp<double>(GCOMPLEX(double)(0.0,1.0)*_stepsize*_pAccessor.template getLink<double>(site)) 
        *_gAcc.template getLink<double>(site);

        temp.su3unitarize();

        return temp;
    }
};

template<class floatT, size_t HaloDepth>
struct do_evolve_P
{
    do_evolve_P(gaugeAccessor<floatT> ipdotAccessor,gaugeAccessor<floatT> pAccessor,floatT stepsize) : _stepsize(stepsize),
    _pAccessor(pAccessor), _ipdotAccessor(ipdotAccessor){}

    floatT _stepsize;
    gaugeAccessor<floatT> _pAccessor;
    gaugeAccessor<floatT> _ipdotAccessor;

    __device__ __host__ GSU3<floatT> operator()(gSiteMu site){
        typedef GIndexer<All,HaloDepth> GInd;

        GSU3<double> temp;

        temp = _pAccessor.template getLink<double>(site);
        temp -= GCOMPLEX(double)(0.0,1.0)*_stepsize *_ipdotAccessor.template getLink<double>(site);

        return temp;
    }
};

template<class floatT, size_t HaloDepth, CompressionType comp=R18>
struct get_gauge_Force
{
    gaugeAccessor<floatT, comp> _gAcc;
    floatT _beta;

    get_gauge_Force(gaugeAccessor<floatT, comp> gAcc, floatT beta) : _gAcc(gAcc), _beta(beta){}

    __device__ __host__ GSU3<floatT> operator()(gSiteMu siteM){
        typedef GIndexer<All,HaloDepth> GInd;
        gSite site(GInd::getSite(siteM.isite));

        GSU3<floatT> temp;
        // temp =- _beta/3.0*symanzikGaugeActionDeriv<floatT,HaloDepth>(_gAcc, site, siteM.mu);
        temp = gauge_force<floatT,HaloDepth,comp>(_gAcc, siteM, _beta);

        return temp;
    }
};


//only for testing 
template<class floatT, size_t HaloDepth>
struct get_mom_tr
{
    gaugeAccessor<floatT> _pAccessor;
    get_mom_tr(gaugeAccessor<floatT> pAccessor): _pAccessor(pAccessor){}

    __device__ __host__ floatT operator()(gSite site){
        typedef GIndexer<All,HaloDepth> GInd;

        floatT ret = 0.0;

        for (int mu = 0; mu < 4; mu++) {
            ret += abs(tr_c(_pAccessor.getLink(GInd::getSiteMu(site, mu))));
        }
        return ret;
    }
};

// this is called from outside, append switch cases if other integration schemes are added
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::integrate(){

    switch(_rhmc_param.integrator())
    {
        case 0:
            SWleapfrog();
            break;

        case 1:
            PQPQP2MN();
            break;

        default:
            rootLogger.error("Unkown integration method!");
    }
}


// Sexton-Weingarten integration scheme
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::SWleapfrog(){

    floatT ieps, iepsh, steph_1f, step_1f, sw_step, sw_steph;

    floatT chmp0=_rhmc_param.mu_f();

    ieps = _rhmc_param.step_size();
    iepsh = 0.5 * ieps;

    step_1f = _rhmc_param.step_size()/_rhmc_param.no_step_1f();
    steph_1f = 0.5* step_1f;

    sw_step = step_1f/_rhmc_param.no_sw();
    sw_steph = 0.5 *sw_step;

    //==================================================//
    // Perform the first half step                      //
    //==================================================//

    updateP_fermforce( iepsh, _phi_2f, true);
    updateP_fermforce( steph_1f, _phi_1f, false);
    
    updateP_gaugeforce( sw_steph );


    rootLogger.info("Done first Leapfrog step");

    //==================================================//
    // Perform the next ( _no_md - 1 ) steps            //
    //==================================================//


    for (int md=1; md<_rhmc_param.no_md(); md++)                  // start loop over steps of 2f
    {
        for (int step=1; step<=_rhmc_param.no_step_1f();step++)   // start loop over steps of 1f
        {
            for (int sw=1; sw<=_rhmc_param.no_sw(); sw++)         // start loop over steps of gauge part
            {
                evolveQ( sw_step );
                updateP_gaugeforce( sw_step );
            }// end loop over steps of gauge part
           if (_rhmc_param.mu_f() !=0){
            _smearing.SmearAll(chmp0);
           }
           else {
               _smearing.SmearAll(); 
           }
            // update P using only the 1f part of the force
            rootLogger.info("strange force:");
            updateP_fermforce( step_1f, _phi_1f, false); 
           
        }// end loop over steps of 1f
        rootLogger.info("light force:");
        // update P using only the 2f part of the force
          updateP_fermforce( ieps, _phi_2f, true); 
       
    }  


    //==================================================//
    // Perform the last half step                       //
    //==================================================// 

    // bring P steph_1f away from the end of the trajectory for 1f part of the force

    for (int step=1; step<_rhmc_param.no_step_1f(); step++)
    {
        for (int sw = 1; sw<=_rhmc_param.no_sw(); sw++)
        {
            evolveQ( sw_step );
            updateP_gaugeforce( sw_step );
        }
        if (_rhmc_param.mu_f() != 0){
             _smearing.SmearAll(chmp0);
        }
        else {
            _smearing.SmearAll();
        }
        updateP_fermforce( step_1f, _phi_1f, false );
    }

    // bring P sw_steph away from the end of the trajectory for gauge part of the force

    for (int sw=1; sw<_rhmc_param.no_sw(); sw++)
    {
        evolveQ( sw_step );
        updateP_gaugeforce( sw_step );
    }
    

    // bring Q to the end of the trajectory

    evolveQ( sw_step );
   if (_rhmc_param.mu_f() != 0){
             _smearing.SmearAll(chmp0);
        }
        else {
            _smearing.SmearAll();
        }

    // bring P to the end of the trajectory by updating with all the forces
    updateP_fermforce( steph_1f, _phi_1f, false); 
    updateP_fermforce( steph_1f, _phi_1f, true); 
    updateP_gaugeforce( sw_steph );
}
// 2MN PQPQP Omelyan integrator on all scales 
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::PQPQP2MN(){

  double ieps, ieps3, ieps6, ieps23;
  double step_1f, step3_1f, step6_1f, step23_1f;
  double sw_step, sw_steph, sw_step3, sw_step6, sw_step23;
  double tauP1, forceP1;
  double tauP2, forceP2;
  double tauQ, tauG, forceG;
  double chmp0;
  double shadow1, shadow2;
  
  ieps = _rhmc_param.step_size();
  ieps3 = ieps/3.0;
  ieps6 = ieps/6.0;
  ieps23 = 2.0*ieps/3.0;
  
  step_1f = _rhmc_param.step_size()/_rhmc_param.no_step_1f()/2.0;
  step3_1f = step_1f/3.0;
  step6_1f = step_1f/6.0;
  step23_1f = 2.0*step_1f/3.0;
  
  sw_step = step_1f/_rhmc_param.no_sw()/2.0;
  sw_steph = sw_step/2.0;
  sw_step3 = sw_step/3.0;
  sw_step6 = sw_step/6.0;
  sw_step23 = 2.0*sw_step/3.0;

  chmp0 = _rhmc_param.mu_f();

  //==================================================//
  // Perform the first half step                      //
  //==================================================//
  
  updateP_fermforce( ieps6, _phi_2f, true);
  forceP2 = forceinfo2();
  tauP2=ieps6;

  updateP_fermforce( step6_1f, _phi_1f, false);    
  forceP1 = forceinfo2();
  tauP1=step6_1f;
  
  updateP_gaugeforce( sw_step6);
  forceG = forceinfo2();
  tauG=sw_step6;

  //calculation of the 2nd order Poisson brackets
  //see 1801.06412
  shadow1=forceP2+(forceP1/pow(2*_rhmc_param.no_step_1f(),2))+(forceG/pow(4*_rhmc_param.no_step_1f()*_rhmc_param.no_sw(),2));

  rootLogger.info( "Done initial P integration step");

  tauQ=0.0;
  //==================================================//
  // Perform the next ( _no_md - 1 ) steps            //
  //==================================================//
  for (int md=1; md< _rhmc_param.no_md(); md++)                  // start loop over steps of 2f
    {
      for (int step=1; step <= _rhmc_param.no_step_1f(); step++)   // start loop over steps of 1f
        {
	  for (int sw=1; sw <= _rhmc_param.no_sw(); sw++)         // start loop over steps of gauge part
            {
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
            }// end loop over steps of gauge part

	  forceinfo();

	  _smearing.SmearAll(chmp0);
	  // update P using only the 1f part of the force
	  rootLogger.info( "strange force:");
	  updateP_fermforce( step23_1f, _phi_1f, false); tauP1+=step23_1f;

	  for (int sw=1; sw <= _rhmc_param.no_sw(); sw++)         // start loop over steps of gauge part
            {
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
            }// end loop over steps of gauge part

	  forceinfo();

	  _smearing.SmearAll(chmp0);
	  // update P using only the 1f part of the force
	  rootLogger.info("strange force:");
	  updateP_fermforce( step3_1f, _phi_1f, false); tauP1+=step3_1f;
        }// end loop over steps of 1f

      rootLogger.info("light force:");
      // update P using only the 2f part of the force
      updateP_fermforce( ieps23, _phi_2f, true); tauP2+=ieps23;

      for (int step=1; step <= _rhmc_param.no_step_1f(); step++)   // start loop over steps of 1f
        {
	  for (int sw=1; sw <= _rhmc_param.no_sw(); sw++)         // start loop over steps of gauge part
            {
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
            }// end loop over steps of gauge part

	  forceinfo();

	  _smearing.SmearAll(chmp0);
	  // update P using only the 1f part of the force
	  rootLogger.info("strange force:");
	  updateP_fermforce( step23_1f, _phi_1f, false); tauP1+=step23_1f;

	  for (int sw=1; sw <= _rhmc_param.no_sw(); sw++)         // start loop over steps of gauge part
            {
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	      evolveQ( sw_steph );             tauQ += sw_steph;
	      updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
            }// end loop over steps of gauge part

	  forceinfo();

	  _smearing.SmearAll(chmp0);
	  // update P using only the 1f part of the force
	  rootLogger.info("strange force:");
	  updateP_fermforce( step3_1f, _phi_1f, false); tauP1+=step3_1f;
        }// end loop over steps of 1f

      rootLogger.info("light force:");
      // update P using only the 2f part of the force
      updateP_fermforce( ieps3, _phi_2f, true); tauP2+=ieps3;
    }  

  //==================================================//
  // Perform the last half step                       //
  //==================================================// 
  // bring P steph_1f away from the end of the trajectory for 1f part of the force
  for (int step=1; step <= _rhmc_param.no_step_1f(); step++)
    {
      for (int sw = 1; sw <= _rhmc_param.no_sw(); sw++)
        {
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
        }

      forceinfo();
      
      _smearing.SmearAll(chmp0);
      rootLogger.info("strange force:");
      updateP_fermforce( step23_1f, _phi_1f, false); tauP1 += step23_1f;

      for (int sw = 1; sw <= _rhmc_param.no_sw(); sw++)
        {
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
        }

      forceinfo();

      _smearing.SmearAll(chmp0);
      rootLogger.info("strange force:");
      updateP_fermforce( step3_1f, _phi_1f, false); tauP1+=step3_1f;
    }
    
  rootLogger.info("light forces:");
  // update P using only the 2f part of the force
  updateP_fermforce( ieps23, _phi_2f, true); tauP2+=ieps23;

  for (int step=1; step < _rhmc_param.no_step_1f(); step++)
    {
      for (int sw = 1; sw <= _rhmc_param.no_sw(); sw++)
        {
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
        }

      forceinfo();
      
      _smearing.SmearAll(chmp0);
      rootLogger.info("strange force:");
      updateP_fermforce( step23_1f, _phi_1f, false); tauP1+=step23_1f;

      for (int sw = 1; sw <= _rhmc_param.no_sw(); sw++)
        {
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
	  evolveQ( sw_steph );             tauQ += sw_steph;
	  updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
        }

      forceinfo();

      _smearing.SmearAll(chmp0);
      rootLogger.info("strange force:");
      updateP_fermforce( step3_1f, _phi_1f, false); tauP1+=step3_1f;
    }

  // bring P sw_steph away from the end of the trajectory for gauge part of the force
  for (int sw=1; sw <= _rhmc_param.no_sw(); sw++)
    {
      evolveQ( sw_steph );             tauQ += sw_steph;
      updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
      evolveQ( sw_steph );             tauQ += sw_steph;
      updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
    }

  forceinfo();

  _smearing.SmearAll(chmp0);
  updateP_fermforce( step23_1f, _phi_1f, false); tauP1+=step23_1f;

  for (int sw=1; sw < _rhmc_param.no_sw(); sw++)
    {
      evolveQ( sw_steph );             tauQ += sw_steph;
      updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
      evolveQ( sw_steph );             tauQ += sw_steph;
      updateP_gaugeforce( sw_step3 );  tauG += sw_step3;
    }

  forceinfo();

  // bring Q to the end of the trajectory
  evolveQ( sw_steph );             tauQ += sw_steph;
  updateP_gaugeforce( sw_step23 ); tauG += sw_step23;
  evolveQ( sw_steph );             tauQ += sw_steph;

  // bring P to the end of the trajectory by updating with all the forces
  updateP_gaugeforce( sw_step6);
  forceG=forceinfo2();
  tauG+=sw_step6;

  _smearing.SmearAll(chmp0);
  updateP_fermforce( step6_1f, _phi_1f, false);
  forceP1=forceinfo2();
  tauP1+=step6_1f;

  updateP_fermforce( ieps6, _phi_2f, true);
  forceP2=forceinfo2();
  tauP2+=ieps6;

  rootLogger.info( "End of trajectory: ");
  rootLogger.info(" ... tauQ  = ", tauQ);
  rootLogger.info(" ... tauG  = ",tauG);
  rootLogger.info(" ... tauP1 = ",tauP1);
  rootLogger.info(" ... tauP2 = ",tauP2);
    
  //calculation of the 2nd order Poisson brackets
  //see 1801.06412
  shadow2=forceP2+(forceP1/pow(2*_rhmc_param.no_step_1f(),2))+(forceG/pow(4*_rhmc_param.no_step_1f()*_rhmc_param.no_sw(),2));
  //Hamiltonian violations to 2nd order
  rootLogger.info("Delta Shadow = ",pow(ieps,2)*(shadow2 - shadow1)/72.0);
}


//update P with the gauge force
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::updateP_gaugeforce(floatT stepsize){

    ipdot.iterateOverBulkAllMu(get_gauge_Force<floatT,HaloDepth,R18>(gAcc, _rhmc_param.beta()));

    evolveP(stepsize);
}

//update P with the fermion force
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::updateP_fermforce(floatT stepsize, 
    Spinorfield<floatT, onDevice, Even, HaloDepthSpin> &phi, bool light/* std::vector<floatT> rat_coeff*/){

  
    ip_dot_f2_hisq.updateForce(phi,ipdot,light);

    forceinfo();

    evolveP(stepsize);
}

template<class floatT, size_t HaloDepth>
struct trace
{

    trace(gaugeAccessor<floatT> ipdotAccessor) : _ipdotAccessor(ipdotAccessor){}

    gaugeAccessor<floatT> _ipdotAccessor;
    

    __device__ __host__ floatT operator()(gSite site){
        typedef GIndexer<All,HaloDepth> GInd;

        GSU3<floatT> temp;

        floatT ret =0.0;

        for(int mu=0; mu<4; mu++)
        {

        temp= _ipdotAccessor.getLink(GInd::getSiteMu(site, mu));

        ret += -2.0 * tr_d(temp,temp);

        }


        return ret;
    }
};

template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::forceinfo(){

    typedef GIndexer<All,HaloDepth> GInd;
    LatticeContainer<onDevice,floatT> force_tr(_p.getComm(), "forcetr");
    force_tr.adjustSize(GInd::getLatData().vol4);

    force_tr.template iterateOverBulk<All, HaloDepth>(trace<floatT, HaloDepth>(ipdotAccessor));

    floatT thing;
    force_tr.reduce(thing, GInd::getLatData().vol4);

    thing = thing /(4* GInd::getLatData().globvol4);



    rootLogger.info("Average force = " ,  thing);


}

template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
floatT integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::forceinfo2(){

    typedef GIndexer<All,HaloDepth> GInd;
    LatticeContainer<onDevice,floatT> force_tr(_p.getComm(), "forcetr");
    force_tr.adjustSize(GInd::getLatData().vol4);

    force_tr.template iterateOverBulk<All, HaloDepth>(trace<floatT, HaloDepth>(ipdotAccessor));

    floatT thing;
    force_tr.reduce(thing, GInd::getLatData().vol4);

    return thing;

}


// update the gauge field
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::evolveQ(floatT stepsize){

    _gaugeField.iterateOverBulkAllMu(do_evolve_Q<floatT, HaloDepth, R18>(gAcc, pAccessor, stepsize));
    _gaugeField.updateAll();
}

//helper function, called in updateP_Xforce
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::evolveP(floatT stepsize){

    _p.iterateOverBulkAllMu(do_evolve_P<floatT, HaloDepth>(ipdotAccessor, pAccessor, stepsize));
}

//test if momenta are traceless, only used in tests
template<class floatT, bool onDevice, Layout LatticeLayout, size_t HaloDepth, size_t HaloDepthSpin>
void integrator<floatT, onDevice, LatticeLayout, HaloDepth, HaloDepthSpin>::check_traceless(){

    typedef GIndexer<All,HaloDepth> GInd;
    LatticeContainer<onDevice,floatT> redBase(_p.getComm());
    const size_t elems = GInd::getLatData().vol4;

    redBase.adjustSize(elems);

    redBase.template iterateOverBulk<All, HaloDepth>(get_mom_tr<floatT, HaloDepth>(pAccessor));

    floatT momenta;

    redBase.reduce(momenta, elems);

    rootLogger.info("summed trace of momenta: " ,  momenta);
}


// this is called from outside, append switch cases if other integration schemes are added
template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
void pure_gauge_integrator<floatT, onDevice, HaloDepth, comp>::integrate(){

    PureGaugeleapfrog();
}


// Leapfrogger for pure gauge HMC
template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
void pure_gauge_integrator<floatT, onDevice, HaloDepth, comp>::PureGaugeleapfrog(){

    //==================================================//
    // Perform the first half step                      //
    //==================================================//

    updateP_gaugeforce( _rhmc_param.step_size()/2.0 );

    rootLogger.info("Done first Leapfrog step");

    //==================================================//
    // Perform the next ( _no_md - 1 ) steps            //
    //==================================================//

    for (int sw=1; sw<_rhmc_param.no_md(); sw++)
    {
        evolveQ( _rhmc_param.step_size() );
        updateP_gaugeforce( _rhmc_param.step_size() );
    }   

    //==================================================//
    // Perform the last half step                       //
    //==================================================// 

    evolveQ(_rhmc_param.step_size());
    updateP_gaugeforce( _rhmc_param.step_size()/2.0 );
}

//update P with the gauge force
template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
void pure_gauge_integrator<floatT, onDevice, HaloDepth, comp>::updateP_gaugeforce(floatT stepsize){

    ipdot.iterateOverBulkAllMu(get_gauge_Force<floatT,HaloDepth,comp>(gAcc, _rhmc_param.beta()));
    evolveP(stepsize);
    // check_traceless();
}


// update the gauge field
template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
void pure_gauge_integrator<floatT, onDevice, HaloDepth, comp>::evolveQ(floatT stepsize){

    _gaugeField.iterateOverBulkAllMu(do_evolve_Q<floatT, HaloDepth, comp>(gAcc, pAccessor, stepsize));
    _gaugeField.updateAll();
}

//helper function, called in updateP_Xforce
template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
void pure_gauge_integrator<floatT, onDevice, HaloDepth, comp>::evolveP(floatT stepsize){

    _p.iterateOverBulkAllMu(do_evolve_P<floatT, HaloDepth>(ipdotAccessor, pAccessor, stepsize));
}

//test if momenta are traceless, only used in tests
template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
void pure_gauge_integrator<floatT, onDevice, HaloDepth, comp>::check_traceless(){

    typedef GIndexer<All,HaloDepth> GInd;
    LatticeContainer<onDevice,floatT> redBase(_p.getComm());
    const size_t elems = GInd::getLatData().vol4;

    redBase.adjustSize(elems);

    redBase.template iterateOverBulk<All, HaloDepth>(get_mom_tr<floatT, HaloDepth>(pAccessor));

    floatT momenta;

    redBase.reduce(momenta, elems);

    rootLogger.info("summed trace of momenta: " ,  momenta);
}


// explicit instantiation
// template class integrator<float, All, 2>;
#define CLASS1_INIT(floatT, HALO, HALOSPIN)			\
template class integrator<floatT, true, All, HALO, HALOSPIN>;
#define CLASS2_INIT(floatT, HALO, comp) \
template class pure_gauge_integrator<floatT, true, HALO, comp>;

INIT_PHHS(CLASS1_INIT)
INIT_PHC(CLASS2_INIT)

